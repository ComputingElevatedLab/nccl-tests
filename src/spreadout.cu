/*************************************************************************
 * Copyright (c) 2016-2022, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "hip/hip_runtime.h"
#include "common.h"

void SpreadoutGetCollByteCount(size_t *sendcount, size_t *recvcount, size_t *paramcount, size_t *sendInplaceOffset, size_t *recvInplaceOffset, size_t count, int nranks)
{
    *sendcount = (count / nranks) * nranks;
    *recvcount = (count / nranks) * nranks;
    *sendInplaceOffset = 0;
    *recvInplaceOffset = 0;
    *paramcount = count / nranks;
}

testResult_t SpreadoutInitData(struct threadArgs *args, ncclDataType_t type, ncclRedOp_t op, int root, int rep, int in_place)
{
    size_t sendcount = args->sendBytes / wordSize(type);
    size_t recvcount = args->expectedBytes / wordSize(type);
    int nranks = args->nProcs * args->nThreads * args->nGpus;

    for (int i = 0; i < args->nGpus; i++)
    {
        CUDACHECK(hipSetDevice(args->gpus[i]));
        int rank = ((args->proc * args->nThreads + args->thread) * args->nGpus + i);
        CUDACHECK(hipMemset(args->recvbuffs[i], 0, args->expectedBytes));
        void *data = in_place ? args->recvbuffs[i] : args->sendbuffs[i];
        TESTCHECK(InitData(data, sendcount, 0, type, ncclSum, 33 * rep + rank, 1, 0));
        for (int j = 0; j < nranks; j++)
        {
            size_t partcount = sendcount / nranks;
            TESTCHECK(InitData((char *)args->expected[i] + j * partcount * wordSize(type), partcount, rank * partcount, type, ncclSum, 33 * rep + j, 1, 0));
        }
        CUDACHECK(hipDeviceSynchronize());
    }
    // We don't support in-place spreadout
    args->reportErrors = in_place ? 0 : 1;
    return testSuccess;
}

void SpreadoutGetBw(size_t count, int typesize, double sec, double *algBw, double *busBw, int nranks)
{
    double baseBw = (double)(count * nranks * typesize) / 1.0E9 / sec;

    *algBw = baseBw;
    double factor = ((double)(nranks - 1)) / ((double)(nranks));
    *busBw = baseBw * factor;
}

testResult_t SpreadoutRunColl(void *sendbuff, void *recvbuff, size_t count, ncclDataType_t type, ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream)
{
    int nRanks, rank;
    NCCLCHECK(ncclCommCount(comm, &nRanks));
    NCCLCHECK(ncclCommUserRank(comm, &rank));
    size_t rankOffset = count * wordSize(type);

#if NCCL_MAJOR < 2 || NCCL_MINOR < 7
    printf("NCCL 2.7 or later is needed for spreadout. This test was compiled with %d.%d.\n", NCCL_MAJOR, NCCL_MINOR);
    return testNcclError;
#else
    ncclGroupStart();
    for (int r = 0; r < nRanks; r++)
    {
        int src = (rank + r) % nRanks;
        ncclRecv(((char *)recvbuff) + (src * rankOffset), count, type, src, comm, stream);
    }
    for (int r = 0; r < nRanks; r++)
    {
        int dst = (rank - r + nRanks) % nRanks;
        ncclSend(((char *)sendbuff) + (dst * rankOffset), count, type, dst, comm, stream);
    }
    ncclGroupEnd();
    return testSuccess;
#endif
}

struct testColl spreadoutTest = {
    "Spreadout",
    SpreadoutGetCollByteCount,
    SpreadoutInitData,
    SpreadoutGetBw,
    SpreadoutRunColl};

void SpreadoutGetBuffSize(size_t *sendcount, size_t *recvcount, size_t count, int nranks)
{
    size_t paramcount, sendInplaceOffset, recvInplaceOffset;
    SpreadoutGetCollByteCount(sendcount, recvcount, &paramcount, &sendInplaceOffset, &recvInplaceOffset, count, nranks);
}

testResult_t SpreadoutRunTest(struct threadArgs *args, int root, ncclDataType_t type, const char *typeName, ncclRedOp_t op, const char *opName)
{
    args->collTest = &spreadoutTest;
    ncclDataType_t *run_types;
    const char **run_typenames;
    int type_count;

    if ((int)type != -1)
    {
        type_count = 1;
        run_types = &type;
        run_typenames = &typeName;
    }
    else
    {
        type_count = test_typenum;
        run_types = test_types;
        run_typenames = test_typenames;
    }

    for (int i = 0; i < type_count; i++)
    {
        TESTCHECK(TimeTest(args, run_types[i], run_typenames[i], (ncclRedOp_t)0, "none", -1));
    }
    return testSuccess;
}

struct testEngine spreadoutEngine = {
    SpreadoutGetBuffSize,
    SpreadoutRunTest};

#pragma weak ncclTestEngine = spreadoutEngine
